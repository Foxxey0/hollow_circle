#define EXPORT __declspec(dllexport)


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void drawPixel(float posX, float posY, float radius, float borderWidth, int width, int height, long pixels, uint8_t *d_colours) {
  uint32_t threadIndex = (threadIdx.z *blockDim.x *blockDim.y) +(threadIdx.y *blockDim.x) +threadIdx.x;
  uint32_t blockIndex = (blockIdx.z *gridDim.x *gridDim.y) +(blockIdx.y *gridDim.x) +blockIdx.x;
  uint32_t index = blockIndex *blockDim.x *blockDim.y *blockDim.z +threadIndex;
  if (index > pixels -1) {
    return;
  }
  uint16_t pixelX = index %width; // TODO
  uint16_t pixelY = index /width;

  float innerRadius = radius -borderWidth +1;
  if (innerRadius > radius) {
    innerRadius = radius;
  }

  float radiusSquared = radius *radius;
  float innerRadiusSquared = innerRadius *innerRadius;

  float distanceX = pixelX -posX;
  float distanceY = pixelY -posY;
  float distanceSquared = (distanceX *distanceX) +(distanceY *distanceY);
  if (distanceSquared < radiusSquared && distanceSquared > innerRadiusSquared) {
    // draw white
    d_colours[index *3 +0] = 255; // red
    d_colours[index *3 +1] = 254; // green
    d_colours[index *3 +2] = 253; // blue
  } else {
    // draw black
    d_colours[index *3 +0] = 100; // red
    d_colours[index *3 +1] = 102; // green
    d_colours[index *3 +2] = 101; // blue
  }
}

extern "C" EXPORT void drawHollowCircleWithBorder(float posX, float posY, float radius, float borderWidth, int width, int height, uint8_t *colours) {
  
  uint8_t *d_colours;
  const uint32_t pixels = width *height;
  const uint32_t bytes = pixels *3 *sizeof(uint8_t);
 
  hipMalloc(&d_colours, bytes);

  struct dim3 gridDim_ = {(pixels /256) +1, 1, 1};
  struct dim3 blockDim_ = {32, 4, 2};
  drawPixel<<<gridDim_, blockDim_>>>(posX, posY, radius, borderWidth, width, height, pixels, d_colours); // 1 thread per pixel

  hipMemcpy(colours, d_colours, bytes, hipMemcpyDeviceToHost); // to, from

  hipFree(d_colours);

  // DEBUG
  // colours[0] = 230;
  // colours[1] = 100;
  // colours[2] = 30;

  return;

}