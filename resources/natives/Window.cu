#define EXPORT __declspec(dllexport)

// #define sqrtOf2 1.4142135623730950488016887242097f
#define halfOfsqrtOf2 0.7071067690849304f

#include <windows.h>
#include <stdio.h>
#include <d3d11.h>
#include <dxgi.h>
#include <cuda_d3d11_interop.h>

extern "C" {
    #include "Init.h"
    #include "MathHelper.h"
}
#include "CudaStuff.h"

// I made my own point struct for mousepos (inside screen, so cant be negative) because i realized i never need anything bigger than a short for screen dimension stuff like mouse position.
struct foxxey_MousePos {
    UINT16 x;
    UINT16 y;
};

struct resizeSafely_threadFunction_parameters {
    INT16 posX;
    INT16 posY;
    UINT16 width;
    UINT16 height;
};

struct threadInfo {
    HANDLE handle;
    DWORD id;
};

HWND windowHandle;
// HDC windowDeviceContextHandle; // its like a windowHandle, but for GDI. So its a graphical interface handle. this acts as a front buffer (but its actually a second buffer since this isnt the screen. theres a step after this thats a real front buffer). whatever is drawn here will be drawn to the window graphics which THEN get composed into the final display thing which includes other windows and stuff.
// BITMAPINFOHEADER bitmapInfoHeader;
// BITMAPINFO bitmapInfo; // contains the bitmapInfoHeader.
// byte *bitmap; // has to be in BGR, and every row must be a multiple of 4 bytes long. just add some bytes of padding at the end of the row to do that.
// HDC memoryDeviceContextHandle; // a separate memory device context. this acts as the second buffer (but its actually the third buffer). we can draw to this one at any time.
// HBITMAP bitmapHandle; // is in context of the memoryDeviceContextHandle. this is the actual bitmap handle for our "second" buffer.

struct threadInfo *threads;
UINT16 threadsArraySize = 4;
UINT16 threadsNextEmptyIndex = 0; // im using this method cuz its smart. every time i add a new thread, i just do it in this index. if the index becomes bigger than the array size, THEN i check for dead threads and take their spots, and if THOSE are full, then increase array size.

// DWORD windowLogic_id;
// HANDLE windowLogic_handle;

// DWORD resizeBitmap_id;
// HANDLE resizeBitmap_handle;

// drag variables
boolean dragging = FALSE;
UINT16 draggingRelativeX = 0; // mouse position relative to window position (top left corner of window). mousePos -windowPos
UINT16 draggingRelativeY = 0;

// resize variables (theyre shorts because thats all i need. pixel dimensions on screen will always fit in shorts)
UINT16 resizeBorder = 20;
boolean resizing = FALSE;
boolean cursorOnResizeBorder = FALSE;
UINT8 resizingType = 0; // clockwise from top left. Ex. 0 - top left, 1 - top, 2 - top right, 3 - right. theres only 8 types. so this value is from 0 to 7.
INT16 resizingOriginalMousePosXRelativeToPos = 0; // its just the way the c window procedure parameters are. the mouse pos is relative to the window pos (top left).
INT16 resizingOriginalMousePosYRelativeToPos = 0;
INT16 resizingOriginalPosX = 0;
INT16 resizingOriginalPosY = 0;
UINT16 resizingOriginalWidth = 0;
UINT16 resizingOriginalHeight = 0;

// fullscreen stuff (custom fullscreen stuff)
BOOL canToggleFullscreen = TRUE;
INT16 fullscreenOriginalPosX = 0;
INT16 fullscreenOriginalPosY = 0;
UINT16 fullscreenOriginalWidth = 200; // just some default values, just in case.
UINT16 fullscreenOriginalHeight = 200;

// for resizeSafely() to work. wait until bitmap is NOT being drawn to, before doing ANYTHING including resizing cuz that would break the bitmap drawing midway through. // - But even then, ALSO wait for setPosition to not be happening
// by the way, the reason resizingSafely exists is so that i can resize the window from another thread. This is needed for the fullscreen thing because its triggered in the windowprocedure method, which runs in a different thread. But the manual resizing doesnt. the manual resizing happens in the same thread as the bitmap drawing, so that doesnt need a safety, its checked first.
BOOL resizingSafely = FALSE;
BOOL drawingToBitmap = FALSE;

// framebuffer stuff
ID3D11Device* devicePointer;
ID3D11DeviceContext* deviceContextPointer;
IDXGISwapChain* swapChainPointer;
ID3D11Texture2D* backBufferPointer;

hipGraphicsResource *cudaResource;
hipArray_t hipArray;

// helper method
// void printBits(byte *bytesPointer, UINT8 bytesAmount) { // READS IT AS IT IS ON THE SYSTEM. if its a known type like short or int or long or anything, then it wont print the way you expect. modern systems use little-endian, meaning the order of BYTES is reversed. the order of bits is the same though.
//     byte currentByte;
//     byte currentBit; // we still have to represent it as a byte since we cant go smaller than that.
//     for (int byteIndex = 0; byteIndex <= bytesAmount -1; byteIndex++) { // loops through bytes
//         currentByte = bytesPointer[byteIndex];
//         for (int bitIndex = 0; bitIndex <= 8 -1; bitIndex++) { // loops through bits
//             currentBit = currentByte & 0b10000000; // get truncated version of byte
//             if (currentBit == 0b10000000) {
//                 printf("1");
//             } else if (currentBit == 0b00000000) {
//                 printf("0");
//             } else {
//                 printf("wrong.");
//             }
//             currentByte = currentByte << 1; // prepares next one
//         }
//     }
//     printf("\n");
// }

// prints the status of each thread (if not null, its alive, i think.)
void threads_print() {
    printf("size: %u, nextempty: %u, ", threadsArraySize, threadsNextEmptyIndex);
    for (int index = 0; index <= threadsArraySize -1; index++ ) {
        if (threads[index].id == NULL) {
            printf("NULL, ");
        } else {
            DWORD threadStatus = WaitForSingleObject(threads[index].handle, 0);
            // if (threadStatus == WAIT_OBJECT_0) { // thread is dead
            //     threadHandles[index] = threadHandle;
            //     return;
            // }
            printf("status%lu, ", threadStatus);
        }
    }
    printf("\n");
}

void foxxey_Exit(INT8 exitCode) { // how can i close all threads before exiting?? i need a thread to exit!. Answer: I made an array of all thread handles that will exist, and i will close every single one thats not the current one, and then with this remaining thread, i do exit.
    DWORD currentThreadId = GetCurrentThreadId();

    for (int index = 1; index <= threadsArraySize -1; index++) { // index 0 is ALWAYS the MAIN THREAD. do NOT terminate that.
        struct threadInfo threadInfo = threads[index];
        printf("thread Id: %lu, current Thread Id: %lu\n", threadInfo.id, currentThreadId); // DEBUG
        if (threadInfo.id != NULL && threadInfo.id != currentThreadId) {
            printf("TERMINATE THREAD\n"); // DEBUG
            TerminateThread(threadInfo.handle, 0);
            CloseHandle(threadInfo.handle);
        }
    }

    // printf("END1\n"); // DEBUG

    // printf("END2\n"); // DEBUG
    // DeleteObject(bitmapHandle); // we created this one
    // DeleteDC(memoryDeviceContextHandle); // we created this one
    // ReleaseDC(windowHandle, windowDeviceContextHandle); // we got this one
    // TODO

    // printf("END3\n"); // DEBUG
    threads_print();

    free(threads); // i malloced

    exit(exitCode);
}

// takes next empty spot. if there isnt one, then look through array for a dead thread to use IT'S spot. if cant find any, then double size of array (i heard of this method online) and then add it.
void threads_add(struct threadInfo threadInfo) { // TEST IF IT WORKS
    if (threadsNextEmptyIndex > threadsArraySize -1) { // no more empty indexes. so now search for dead threads so we can take their place with this new thread. if not, then increase array size.
        for (int index = 0; index <= threadsArraySize -1; index++) { // check for dead thread
            DWORD threadStatus = WaitForSingleObject(threads[index].handle, 0);
            if (threadStatus == WAIT_OBJECT_0) { // thread is dead
                threads[index] = threadInfo;
                return;
            }
        }
        // a dead thread wasnt found. all spots are taken by live threads. so lets double the array size!
        UINT16 threadsArraySize_temp = threadsArraySize << 2; // doubles array size variable by 2. i just wanna use bitwise operator cuz its cool! also to make it faster JUST IN CASE it doesnt do it for me if i do *= 2;
        struct threadInfo *threads_temp = (struct threadInfo *) malloc(threadsArraySize_temp *sizeof(struct threadInfo)); // make temporary array of the new size
        for (int index = 0; index <= threadsArraySize -1; index++) { // copy everything from original array to new array.
            threads_temp[index] = threads[index];
        }
        free(threads); // free original array. we dont need it anymore. we have all the thread handles copied to the new array by now.
        threads = threads_temp; // set the array pointer to the temp (new) one.
        threadsArraySize = threadsArraySize_temp; // set the array size variable to the temp (new) one.
        // set all thread ids to NULL just in case.
        for (int index = 0; index <= threadsArraySize -1; index++) {
            threads[index].id = NULL;
        }
    } // else { // an empty index exists! use it immediately. now we dont have to iterate through the array!}

    threads[threadsNextEmptyIndex] = threadInfo;
    threadsNextEmptyIndex++;
}

struct foxxey_MousePos getMousePosFromlParameter(LPARAM lParameter) {
    UINT16 mouseX = (UINT16) lParameter; // lower short (rightmost). mousePos RELATIVE TO WINDOW.
    UINT16 mouseY = (UINT16) (lParameter >> (sizeof(UINT16) *8)); // next short (rightmost). reads lParameter with it shifted to the right to access the left, which are the higher bits. mousePos RELATIVE TO WINDOW. its *8 because im BITshifting, im not BYTEshifting. sizeof() returns amount of bytes.
    struct foxxey_MousePos mousePos = {mouseX, mouseY};
    return mousePos;
}

void setStartResizeVariables(struct foxxey_MousePos mousePos, RECT windowRect) { // variables to set when starting a resize.
    resizingOriginalMousePosXRelativeToPos = mousePos.x;
    resizingOriginalMousePosYRelativeToPos = mousePos.y;
    resizingOriginalPosX = windowRect.left;
    resizingOriginalPosY = windowRect.top;
    resizingOriginalWidth = windowRect.right -windowRect.left;
    resizingOriginalHeight = windowRect.bottom -windowRect.top;
    resizing = TRUE; // here last for protection. to make sure all other variable are set first.
}

// specifically for RGB bitmap
__host__ __device__ long rowBytesAmountToPaddedRowBytesAmount(long rowBytesAmount) {
    UINT8 paddingBytesAmount = 4 -(rowBytesAmount %4);
    if (paddingBytesAmount == 4) {
        paddingBytesAmount = 0;
    }
    return rowBytesAmount +paddingBytesAmount;
}

void resizeBitmap() {
    RECT windowRect;
    if (!GetWindowRect(windowHandle, &windowRect)) {
        printf("EXITING: GetWindowRect() failed.\n");
        foxxey_Exit(-1);
    }
    UINT16 windowWidth = windowRect.right -windowRect.left; // useful to make variable since i use these twice below
    UINT16 windowHeight = windowRect.bottom -windowRect.top; // useful to make variable since i use these twice below

    // free(bitmap);
    // bitmap = (byte *) malloc(rowBytesAmountToPaddedRowBytesAmount(windowWidth *3) *windowHeight);
    // bitmapInfoHeader.biWidth = windowWidth;
    // bitmapInfoHeader.biHeight = -windowHeight;
    // bitmapInfo.bmiHeader = bitmapInfoHeader;
    // TODO
}

// thread function
DWORD WINAPI resizeSafely_threadFunction(LPVOID lpThreadParameter) {

    struct resizeSafely_threadFunction_parameters parameters = *((struct resizeSafely_threadFunction_parameters *) lpThreadParameter); // deference the struct pointer
    // printf("P3width: %ld, P3Height: %ld\n", parameters.width, parameters.height); // DEBUG
    // wait for this function's turn.
    while (1) {
        if (!drawingToBitmap) { // dont resizeSafely while its drawing to bitmap
            resizingSafely = TRUE;
            break;
        }
    }

    if (!SetWindowPos(windowHandle, NULL, parameters.posX, parameters.posY, parameters.width, parameters.height, SWP_NOZORDER)) {
        printf("EXITING: SetWindowPos() failed.\n");
        foxxey_Exit(-1);
    }

    resizeBitmap();

    resizingSafely = FALSE;

    return 0;
}

// resize, then reset bitmap and bitmap info. if resizing in another thread, we dont want it to resize at the same time bitmap is being drawn to. so we have to check the debounce variable and wait for it to be time to resize. but i dont wanna wait in the current thread, so we gotta do it in a new thread. thats what this does.
void resizeSafely(INT16 posX, INT16 posY, UINT16 width, UINT16 height) {
    // thread to resize safely and quickly so we dont have to wait in the current thread. we can do the waiting in this new thread!
    struct resizeSafely_threadFunction_parameters *arguments = (struct resizeSafely_threadFunction_parameters *) malloc(sizeof(struct resizeSafely_threadFunction_parameters)); // must malloc because im gonna pass this to a new thread which a different scope. if i use normal definition, then it will just dissapear as soon as this function ends, and the new thread would have nothing to work with. so you GOTTA malloc it so its on the heap and can be retrieved by ANY scope.
    (*arguments) = {posX, posY, width, height};
    HANDLE resizeSafely_threadHandle;
    DWORD resizeSafely_threadId;
    resizeSafely_threadHandle = CreateThread(NULL, 0, &resizeSafely_threadFunction, arguments, 0, &resizeSafely_threadId);
    struct threadInfo resizeSafely_threadInfo = {resizeSafely_threadHandle, resizeSafely_threadId};
    threads_add(resizeSafely_threadInfo);
    if (resizeSafely_threadInfo.id == NULL) {
        printf("EXITING: resizeSafely_threadInfo.id creation failed.\n");
        foxxey_Exit(-1);
    }
}

// windowHandle is the current window
LRESULT CALLBACK windowProcedure(HWND windowHandle, UINT message, WPARAM wParameter, LPARAM lParameter) {
    // printf("windowHandle: %d, message: %d, wParameter: %d, lParameter: %d\n", windowHandle, message, wParameter, lParameter); // DEBUG

    if (message == WM_MOUSEMOVE) {
        struct foxxey_MousePos mousePos = getMousePosFromlParameter(lParameter);

        RECT windowRect;
        if (!GetWindowRect(windowHandle, &windowRect)) {
            printf("EXITING: GetWindowRect() failed.\n");
            foxxey_Exit(-1);
        }

        if (!resizing) { // check if not resizing. if it is resizing, these variables should not change because the type of resize cant change during a resize.
            cursorOnResizeBorder = TRUE;

            if (mousePos.x <= resizeBorder -1) { // left
                if (mousePos.y <= resizeBorder -1) { // left +top
                    resizingType = 0;
                } else if (mousePos.y >= (windowRect.bottom -windowRect.top) -(resizeBorder -1)) { // left +bottom
                    resizingType = 6;
                } else { // only left
                    resizingType = 7;
                }
            } else if (mousePos.x >= (windowRect.right -windowRect.left) -(resizeBorder -1)) { // right
                if (mousePos.y <= resizeBorder -1) { // right +top
                    resizingType = 2;
                } else if (mousePos.y >= (windowRect.bottom -windowRect.top) -(resizeBorder -1)) { // right +bottom
                    resizingType = 4;
                } else { // only right
                    resizingType = 3;
                }
            } else if (mousePos.y <= resizeBorder -1) { // top
                resizingType = 1;
            } else if (mousePos.y >= (windowRect.bottom -windowRect.top) -(resizeBorder -1)) { // bottom
                resizingType = 5;
            } else {
                cursorOnResizeBorder = FALSE;
            }
        }
    } else if (message == WM_LBUTTONDOWN) { // check resizing first because its on top of the window where you could normally drag.
        if (!dragging) { // safe to resize.
            if (!resizing) { // not resizing yet but can. so START resizing.
                struct foxxey_MousePos mousePos = getMousePosFromlParameter(lParameter);

                RECT windowRect;
                if (!GetWindowRect(windowHandle, &windowRect)) {
                    printf("EXITING: GetWindowRect() failed.\n");
                    foxxey_Exit(-1);
                }
                if (cursorOnResizeBorder) {
                    setStartResizeVariables(mousePos, windowRect);
                }
            }
        }
        if (!resizing) { // safe to drag.
            if (!dragging) { // not dragging yet but can. so START dragging.
                struct foxxey_MousePos mousePos = getMousePosFromlParameter(lParameter);
                dragging = TRUE;
                draggingRelativeX = mousePos.x; // these are already relative to the window so i can set them directly.
                draggingRelativeY = mousePos.y;
                // printf("draggingRelativeX: %hd, draggingRelativeY: %hd\n", draggingRelativeX, draggingRelativeY); // DEBUG
            }
        }
    } else if (message == WM_SETCURSOR) { // cursor stuff. this is called whenever mouse moves over window. and thats fine. thats all i need.
        HCURSOR cursorHandle;
        if (cursorOnResizeBorder) {
            if (resizingType == 0 || resizingType == 4) {
                cursorHandle = LoadCursor(NULL, IDC_SIZENWSE);
            } else if (resizingType == 1 || resizingType == 5) {
                cursorHandle = LoadCursor(NULL, IDC_SIZENS);
            } else if (resizingType == 2 || resizingType == 6) {
                cursorHandle = LoadCursor(NULL, IDC_SIZENESW);
            } else if (resizingType == 3 || resizingType == 7) {
                cursorHandle = LoadCursor(NULL, IDC_SIZEWE);
            }
        } else {
            cursorHandle = LoadCursor(NULL, IDC_ARROW); // default
        }
        if (cursorHandle == NULL) {
            printf("EXITING: LoadCursor() failed.\n");
            foxxey_Exit(-1);
        }
        SetCursor(cursorHandle);
        return TRUE;
    } else if (message == WM_KEYDOWN) {
        // test if F is down to toggle fullscreen
        if (wParameter == 'F') {
            if (canToggleFullscreen) {
                canToggleFullscreen = FALSE;

                RECT windowRect;
                if (!GetWindowRect(windowHandle, &windowRect)) {
                    printf("EXITING: GetWindowRect() failed.\n");
                    foxxey_Exit(-1);
                }

                // printf("Aleft: %ld, Aright: %ld\n", windowRect.left, windowRect.right); // DEBUG

                UINT16 windowWidth = windowRect.right -windowRect.left;
                UINT16 windowHeight = windowRect.bottom -windowRect.top;
                int screenWidth = GetSystemMetrics(SM_CXSCREEN);
                int screenHeight = GetSystemMetrics(SM_CYSCREEN);

                // check if already in fullscreen. if so, then revert to what it was before the toggle fullscreen on.
                if (windowWidth == screenWidth && windowHeight == screenHeight) {
                    printf("turning fullscreen OFF\n"); // DEBUG
                    resizeSafely(fullscreenOriginalPosX, fullscreenOriginalPosY, fullscreenOriginalWidth, fullscreenOriginalHeight);
                    // if (!SetWindowPos(windowHandle, NULL, fullscreenOriginalPosX, fullscreenOriginalPosY, fullscreenOriginalWidth, fullscreenOriginalHeight, SWP_NOZORDER)) {
                    //     printf("EXITING: SetWindowPos() failed.\n");
                    //     foxxey_Exit(-1);
                    // }
                } else { // MAKE it fullscreen and save the original dimension values
                    printf("turning fullscreen ON\n"); // DEBUG
                    fullscreenOriginalPosX = windowRect.left;
                    fullscreenOriginalPosY = windowRect.top;
                    fullscreenOriginalWidth = windowWidth;
                    fullscreenOriginalHeight = windowHeight;
                    resizeSafely(0, 0, screenWidth, screenHeight);
                    // if (!SetWindowPos(windowHandle, NULL, 0, 0, screenWidth, screenHeight, SWP_NOZORDER)) {
                    //     printf("EXITING: SetWindowPos() failed.\n");
                    //     foxxey_Exit(-1);
                    // }
                }
            }
        }
    } else if (message == WM_KEYUP) {
        // test if F is UP to reset canToggleFullscreen debounce
        if (wParameter == 'F') {
            canToggleFullscreen = TRUE;
        }
    }

    return DefWindowProc(windowHandle, message, wParameter, lParameter);
}

// function for a thread
DWORD WINAPI windowLogic_threadFunction(LPVOID lpThreadParameter) {
    SYSTEMTIME startTime; // time how long an iteration takes :)
    SYSTEMTIME currentTime;
    UINT32 timeTaken;
    // float FPS;
    GetLocalTime(&startTime);

    UINT16 counter_cycles = 0;
    while (1) { // do wnidow stuff

        GetLocalTime(&currentTime);
        timeTaken = ((currentTime.wMinute *60 *1000) +(currentTime.wSecond *1000) +(currentTime.wMilliseconds)) -((startTime.wMinute *60 *1000) +(startTime.wSecond *1000) +(startTime.wMilliseconds)); // i dont think i need more than an hour of duration for now, so ill only go up to minutes :)
        // printf("%d\n", timeTaken); // DEBUG
        if (timeTaken >= 1000) { // if 1 second has passed, see how many frames happened in that second
            printf("FPS: %hu\n", counter_cycles);
            counter_cycles = 0; // reset counter
            GetLocalTime(&startTime); // reset timer
        }

        // test if escape is down to exit
        if (GetAsyncKeyState(VK_ESCAPE) == (SHORT) 0b1000000000000000) { // have to add (SHORT) to convert it to a system number which uses little endian. this way it will reverse the order of bytes of this hard-coded thing, which is the order the system reads in, so it will read correctly.
            printf("EXITING: Escape pressed.\n");
            foxxey_Exit(0);
        }

        // test if left mouse button is up to cancel dragging and resizing
        if (GetAsyncKeyState(VK_LBUTTON) == (SHORT) 0b0000000000000000) {
            resizing = FALSE;
            dragging = FALSE;
        }

        if (resizing) { // resizing, so change window size
            POINT mousePos;
            if (GetCursorPos(&mousePos) == 0) {
                printf("EXITING: GetCursorPos() failed.\n");
                foxxey_Exit(-1);
            }
            // i wish i didnt have to use "int" for these. screenwidth even for high-end computers is never over 65,000. and the type that goes up to 65,000 is a UINT16. so UINT16 is more than enough. but windows doesnt accept that, so i just have to pass in an int. stupid windows.
            int mousePosX_delta = mousePos.x -(resizingOriginalMousePosXRelativeToPos +resizingOriginalPosX);
            int mousePosY_delta = mousePos.y -(resizingOriginalMousePosYRelativeToPos +resizingOriginalPosY);

            if (resizingType == 7) { // W
                if (!SetWindowPos(windowHandle, NULL, resizingOriginalPosX +mousePosX_delta, resizingOriginalPosY, resizingOriginalWidth -mousePosX_delta, resizingOriginalHeight, SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 3) { // E
                if (!SetWindowPos(windowHandle, NULL, 0, 0, resizingOriginalWidth +mousePosX_delta, resizingOriginalHeight, SWP_NOMOVE | SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 1) { // N
                if (!SetWindowPos(windowHandle, NULL, resizingOriginalPosX, resizingOriginalPosY +mousePosY_delta, resizingOriginalWidth, resizingOriginalHeight -mousePosY_delta, SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 5) { // S
                if (!SetWindowPos(windowHandle, NULL, 0, 0, resizingOriginalWidth, resizingOriginalHeight +mousePosY_delta, SWP_NOMOVE | SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 0) { // NW
                if (!SetWindowPos(windowHandle, NULL, resizingOriginalPosX +mousePosX_delta, resizingOriginalPosY +mousePosY_delta, resizingOriginalWidth -mousePosX_delta, resizingOriginalHeight -mousePosY_delta, SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 2) { // NE
                if (!SetWindowPos(windowHandle, NULL, resizingOriginalPosX, resizingOriginalPosY +mousePosY_delta, resizingOriginalWidth +mousePosX_delta, resizingOriginalHeight -mousePosY_delta, SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 6) { // SW
                if (!SetWindowPos(windowHandle, NULL, resizingOriginalPosX +mousePosX_delta, resizingOriginalPosY, resizingOriginalWidth -mousePosX_delta, resizingOriginalHeight +mousePosY_delta, SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            } else if (resizingType == 4) { // SE
                if (!SetWindowPos(windowHandle, NULL, 0, 0, resizingOriginalWidth +mousePosX_delta, resizingOriginalHeight +mousePosY_delta, SWP_NOMOVE | SWP_NOZORDER)) {
                    printf("EXITING: SetWindowPos() failed.\n");
                    foxxey_Exit(-1);
                }
            }
            resizeBitmap();
        } else if (dragging) { // dragging, so move window. "else if" because it should only be 1 at a time. protection.
            POINT mousePos;
            if (GetCursorPos(&mousePos) == 0) {
                printf("EXITING: GetCursorPos() failed.\n");
                foxxey_Exit(-1);
            }
            // printf("mouseX: %ld, mouseY: %ld\n", mousePos.x, mousePos.y); // DEBUG
            if (!SetWindowPos(windowHandle, NULL, mousePos.x -draggingRelativeX -1, mousePos.y -draggingRelativeY -1, 0, 0, SWP_NOSIZE | SWP_NOZORDER)) {
                printf("EXITING: SetWindowPos() failed.\n");
                foxxey_Exit(-1);
            }
        }
        
        // draw stuff
        // wait for this function's turn
        while (1) {
            if (!resizingSafely) { // dont draw while its resizingSafely
                drawingToBitmap = TRUE;
                break;
            }
        }

        // Rectangle(windowDeviceContext, 10, 30, 60, 80); // test
        RECT windowRect;
        if (!GetWindowRect(windowHandle, &windowRect)) {
            printf("EXITING: GetWindowRect() failed.\n");
            foxxey_Exit(-1);
        }
        
        // printf("left: %ld, right: %ld\n", windowRect.left, windowRect.right); // DEBUG

        // functional test!
        UINT16 windowWidth = windowRect.right -windowRect.left;
        UINT16 windowHeight = windowRect.bottom -windowRect.top;

        UINT32 windowWidthReciprocal = ((UINT64) 1 << 32) /windowWidth; // gpu tricks! its shifted 32 bits to the left so we can avoid float division. we can just multiply using shifted reciprocal, then shift to the right 32 to get rid of the values that WOULD be less than 1.

        UINT32 pixelsAmount = windowWidth *windowHeight;
        UINT32 paddedRowBytesAmount = rowBytesAmountToPaddedRowBytesAmount(windowWidth *3); // the reason im doing this separate from bytesAmount is cuz the kernals need this value. and of course, i only need to calculate it once, so i might as well calculate it on the cpu.
        UINT32 bytesAmount = paddedRowBytesAmount *windowHeight;
        dim3 gridDim_ = {((pixelsAmount -1) /256) +1, 1, 1};
        dim3 blockDim_ = {32, 4, 2}; // 256 total

        // circle stuff
        float circleMiddleX = windowWidth *.5f; // middle of circle
        float circleMiddleY = windowHeight *.5f; // middle of circle
        float borderWidth = 10;
        UINT32 totalAnimationTime = 9000; // in millis
        UINT32 totalColourAnimationTime = 5000; // in millis

        float radius;
        float progress = getProgress(totalAnimationTime);
        if (progress < .5f) { // 0 to .5
            radius = interpolate_circular_slowFastSlow(progress *2, 10, 500);
        } else { // .5 to 1
            radius = interpolate_circular_slowFastSlow((progress -.5f) *2, 500, 10);
        }

        float innerRadius = radius -borderWidth;

        float radiusSquared = radius *radius;
        float innerRadiusSquared = innerRadius *innerRadius;

        // float innerAABBHalfLength = radius *sqrtOf2 *.5f;
        float innerAABBHalfLength = innerRadius *halfOfsqrtOf2;

        // colour animation
        float colourProgress = getProgress(totalColourAnimationTime);
        struct colour_RGB circleColour = HSBToRGB(colourProgress, 1, 1);

        struct colour_RGB backgroundColour = {40, 43, 48};

        // byte *d_bitmap; // pointer of bitmap on device memory. gpu's version of bitmap.

        hipGraphicsMapResources(1, &cudaResource, 0);
        hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = hipArray;

        hipSurfaceObject_t surface;
        hipCreateSurfaceObject(&surface, &resDesc);

        clearWindow_kernel<<<gridDim_, blockDim_>>>(surface, windowWidth, windowWidthReciprocal, windowHeight, pixelsAmount, paddedRowBytesAmount, backgroundColour); // 1 thread per pixel
        drawHollowCircleWithBorder_kernel<<<gridDim_, blockDim_>>>(surface, windowWidth, windowWidthReciprocal, windowHeight, pixelsAmount, paddedRowBytesAmount, circleMiddleX, circleMiddleY, radiusSquared, innerRadiusSquared, radius, innerAABBHalfLength, circleColour); // 1 thread per pixel
    
        hipDestroySurfaceObject(surface);
        hipGraphicsUnmapResources(1, &cudaResource, 0);

        (*swapChainPointer).Present(0, 0); // why should i need these if i only have 1 buffer??

        drawingToBitmap = FALSE;

        // GetLocalTime(&endTime);
        // timeTaken = endTime.wMilliseconds -startTime.wMilliseconds;
        // FPS = (timeTaken /((float) 1000));
        // if (FPS == 0) { // protection against dividing by 0
        //     printf("MS/F: %u, FPS: INFINITE\n", timeTaken);
        // } else {
        //     FPS = 1 /FPS; // finish the conversion from Milliseconds/Frame to Frames/Second. specifically this line converts it from Seconds/Frame to Frames/Second.
        //     printf("MS/F: %u, FPS: %f\n", timeTaken, FPS);
        // }
        counter_cycles++;
    }
    return 0;
}

// entry point
extern "C" EXPORT void startWindow(void) {

    // init array using malloc
    threads = (struct threadInfo *) malloc(threadsArraySize *sizeof(struct threadInfo));
    // set all thread ids to NULL just in case.
    for (int index = 0; index <= threadsArraySize -1; index++) {
        threads[index].id = NULL;
    }

    HANDLE currentThreadHandle;
    if (!DuplicateHandle(
        GetCurrentProcess(),     // source process
        GetCurrentThread(),           // pseudo-handle
        GetCurrentProcess(),     // target process
        &currentThreadHandle,             // out: real handle
        0,                       // access (ignored because DUPLICATE_SAME_ACCESS)
        FALSE,                   // not inheritable
        DUPLICATE_SAME_ACCESS
    )) {
        printf("CANT EXIT, ERROR: DuplicateHandle() failed.\n");
        exit(-1);
        // foxxey_Exit(-1);
    }
    struct threadInfo currentThreadInfo = {currentThreadHandle, GetCurrentThreadId()};
    threads_add(currentThreadInfo); // main thread's handle gets stored >:)

    printf("Main Thread Id: %lu\n", currentThreadInfo.id); // DEBUG


    // setup swapchain stuff
    DXGI_MODE_DESC bufferDescription = {
        200,
        200,
        {0, 0}, // will tell DXGI to automatically get screen's refresh rate.
        DXGI_FORMAT_R8G8B8A8_UNORM, // store as normal colour values that im used to, but will automatically transform to 0-1, for the screen to display them.
        DXGI_MODE_SCANLINE_ORDER_UNSPECIFIED,
        DXGI_MODE_SCALING_UNSPECIFIED
    };

    DXGI_SAMPLE_DESC sampleDescription = {
        1,
        NULL
    };

    DXGI_SWAP_CHAIN_DESC swapChainDescription = {
        bufferDescription,
        sampleDescription,
        DXGI_USAGE_RENDER_TARGET_OUTPUT,
        1, // 1 buffer.
        windowHandle,
        TRUE,
        DXGI_SWAP_EFFECT_FLIP_DISCARD, // IMPORTANT. this flips (switches pointers), AND doesnt put in effort to save the backbuffer contents. its THE fastest.
        0 | DXGI_SWAP_CHAIN_FLAG_ALLOW_TEARING
    };

    D3D11CreateDeviceAndSwapChain(
        NULL,
        D3D_DRIVER_TYPE_HARDWARE,
        NULL,
        0,
        NULL,
        0,
        D3D11_SDK_VERSION,
        &swapChainDescription,
        &swapChainPointer,
        &devicePointer,
        NULL,
        &deviceContextPointer
    );

    (*swapChainPointer).GetBuffer(0, __uuidof(ID3D11Texture2D), (void **) &backBufferPointer); // gets the pointer to the back buffer so i can draw to it.

    cudaGraphicsD3D11RegisterResource(&cudaResource, backBufferPointer, hipGraphicsRegisterFlagsNone);

    // window stuff
    LPCSTR className = "foxxey_TESTPROGRAM";

    WNDCLASSEX windowClass = {
        sizeof(WNDCLASSEX),
        0,
        windowProcedure, // callback function. handle messages sent to this window. behavior.
        0,
        0,
        DLLHandleInstance, // handle instance of place that has the window procedure (this place) (if its a dll, which this is, it has its own handle instance separate from the calling program. this DLL's handle instance is in DLLHandle)
        NULL, // icon
        NULL, // cursor
        NULL, // background brush
        NULL, // menu name?
        className,
        NULL
    };

    RegisterClassEx(&windowClass);

    if (GetClassInfoEx(DLLHandleInstance, className, &windowClass) != 0) {
        // printf("success: window class success.\n");
    } else {
        // Class not registered or an error occurred
        // You can call GetLastError() for more specific error information
        printf("ERROR: window class failed.\n");
        foxxey_Exit(-1);
    }

    windowHandle = CreateWindowEx(
        // WS_EX_COMPOSITED | WS_EX_LAYERED | WS_EX_NOREDIRECTIONBITMAP | WS_EX_TOPMOST | WS_EX_TRANSPARENT, // OO INTERESTING. LOOK INTO.
        0, // OO INTERESTING. LOOK INTO.
        className,
        NULL, // i dont want it to have a title bar at all, so i dont need a title name.
        WS_VISIBLE | WS_POPUP,
        200, // but it actually starts at x 201
        200, // but it actually starts at y 201
        200,
        200,
        NULL, // parent. this window will be a normal one, so theres no parent.
        NULL, // menu?
        DLLHandleInstance, // handle instance
        NULL
    );

    if (windowHandle == NULL) {
        printf("ERROR: window is null\n");
        foxxey_Exit(-1);
    }


    // windowDeviceContextHandle = GetDC(windowHandle);
    // if (windowDeviceContextHandle == NULL) {
    //     printf("EXITING: GetDC() failed.\n");
    //     foxxey_Exit(-1);
    // }
    // TODO maybe

    // bitmap = (byte *) malloc(rowBytesAmountToPaddedRowBytesAmount(200 *3) *200); // this is doing bytes per row *height. i just think its the best way to do the padding since the padding must be tacked onto the end of every row.

    // bitmapInfoHeader.biSize = sizeof(BITMAPINFOHEADER);
    // bitmapInfoHeader.biWidth = 200;
    // bitmapInfoHeader.biHeight = -200; // must be negative to flip it the right way. its just windows being stupid. real height cant be negative, but apparently setting this height to negative works.
    // bitmapInfoHeader.biPlanes = 1;
    // bitmapInfoHeader.biBitCount = 3 *8;
    // bitmapInfoHeader.biCompression = BI_RGB;
    // bitmapInfoHeader.biSizeImage = 0;
    // bitmapInfoHeader.biXPelsPerMeter = 0; // meta data stuff. doesnt matter.
    // bitmapInfoHeader.biYPelsPerMeter = 0; // meta data stuff. doesnt matter.
    // bitmapInfoHeader.biClrUsed = 0; // no colour table. we want full colour range. a pallette for full-range would be 256^3 colours big. not worth it, clearly. we are better of not using indexes and just setting R G and B in a byte each. 3 bytes per pixel.
    // bitmapInfoHeader.biClrImportant = 0;

    // bitmapInfo.bmiHeader = bitmapInfoHeader; // NOTE: since this uses the value of the struct, not a reference, it must be reset every time bitmapInfoheader changes.
    // bitmapInfo.bmiColors = NULL;

    // memoryDeviceContextHandle = CreateCompatibleDC(windowDeviceContextHandle); // memory
    // if (memoryDeviceContextHandle == NULL) {
    //     printf("EXITING: CreateCompatibleDC() failed.\n");
    //     foxxey_Exit(-1);
    // }
    // bitmapHandle = CreateCompatibleBitmap(windowDeviceContextHandle, 200, 200); // why window device context handle? the docs say to use the HDC that was used to create the memory device context handle so that it uses the right colour scheme.
    // if (bitmapHandle == NULL) {
    //     printf("EXITING: CreateCompatibleBitmap() failed.\n");
    //     foxxey_Exit(-1);
    // }
    // SelectObject(memoryDeviceContextHandle, bitmapHandle);
    
    // SetWindowLong(windowHandle, GWL_STYLE, 0); // removes border?

    // thread to do window logic
    HANDLE windowLogic_threadHandle;
    DWORD windowLogic_threadId;
    windowLogic_threadHandle = CreateThread(NULL, 0, &windowLogic_threadFunction, NULL, 0, &windowLogic_threadId);
    struct threadInfo windowLogic_threadInfo = {windowLogic_threadHandle, windowLogic_threadId};
    printf("windowLogic Thread Id: %lu\n", windowLogic_threadId); // DEBUG
    threads_add(windowLogic_threadInfo);
    if (windowLogic_threadInfo.id == NULL) {
        printf("EXITING: windowLogic_threadInfo.id creation failed.\n");
        foxxey_Exit(-1);
    }

    // window message listener. it HAS to be in the same thread that created the window. this cant in another thread.
    MSG msg;
    while (GetMessage(&msg, windowHandle, NULL, NULL) > 0) {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
    }

    return;
}
