#include "hip/hip_runtime.h"
#include <stdio.h>
#include <windows.h>

extern "C" {
  #include "MathHelper.h"
}

// GPU kernel
__global__ void drawHollowCircleWithBorder_kernel(hipSurfaceObject_t const surface, const UINT16 windowWidth, const UINT32 windowWidthReciprocal, const UINT16 windowHeight, const UINT32 pixelsAmount, const UINT32 paddedRowBytesAmount, const float circleMiddleX, const float circleMiddleY, const float radiusSquared, const float innerRadiusSquared, const float outerAABBHalfLength, const float innerAABBHalfLength, const struct colour_RGB rgb) {
  UINT32 threadIndex = (threadIdx.z *blockDim.x *blockDim.y) +(threadIdx.y *blockDim.x) +threadIdx.x;
  UINT32 blockIndex = (blockIdx.z *gridDim.x *gridDim.y) +(blockIdx.y *gridDim.x) +blockIdx.x;
  UINT32 index = (blockIndex *256) +threadIndex;
  if (index > pixelsAmount -1) {
    return;
  }
  // UINT16 pixelX = index %windowWidth;
  // UINT16 pixelY = index /windowWidth;
  UINT16 pixelY = __umulhi(index ,windowWidthReciprocal); // x *y but returns only high 32 bits, and then get the 16 bits out of the result that i need.
  UINT16 pixelX = index -(pixelY *windowWidth);

  float distanceX = pixelX -circleMiddleX;
  float distanceY = pixelY -circleMiddleY;
  // float distanceX = fabs(pixelX -circleMiddleX);
  // float distanceY = fabs(pixelY -circleMiddleY);

  // AABB check, sort of. its faster than full AABB check. this check PROBABLY makes many warps fail faster than they would have ended if they just did the distanceSquared calculation, saving time.
  // if (distanceX > outerAABBHalfLength || distanceY > outerAABBHalfLength || (distanceX < innerAABBHalfLength && distanceY < innerAABBHalfLength)) {
  //   return;
  // }
  if (distanceX < -outerAABBHalfLength || distanceX > outerAABBHalfLength || distanceY < -outerAABBHalfLength || distanceY > outerAABBHalfLength || (distanceX > -innerAABBHalfLength && distanceX < innerAABBHalfLength && distanceY > -innerAABBHalfLength && distanceY < innerAABBHalfLength)) {
    return;
  }

  float distanceSquared = (distanceX *distanceX) +(distanceY *distanceY);
  if (distanceSquared < radiusSquared && distanceSquared > innerRadiusSquared) {
    // draw white
    // long pixelIndex = (pixelY *paddedRowBytesAmount) +(pixelX *3);
    // d_bitmap[pixelIndex +0] = rgb.blue; // B
    // d_bitmap[pixelIndex +1] = rgb.green; // G
    // d_bitmap[pixelIndex +2] = rgb.red; // R

    uchar4 color = make_uchar4(rgb.red, rgb.green, rgb.blue, 255);
    surf2Dwrite(color, surface, pixelX *sizeof(uchar4), pixelY);
  }
}

// GPU kernel
__global__ void clearWindow_kernel(hipSurfaceObject_t const surface, const UINT16 windowWidth, const UINT32 windowWidthReciprocal, const UINT16 windowHeight, const UINT32 pixelsAmount, const UINT32 paddedRowBytesAmount, const struct colour_RGB rgb) {
  UINT32 threadIndex = (threadIdx.z *blockDim.x *blockDim.y) +(threadIdx.y *blockDim.x) +threadIdx.x;
  UINT32 blockIndex = (blockIdx.z *gridDim.x *gridDim.y) +(blockIdx.y *gridDim.x) +blockIdx.x;
  UINT32 index = (blockIndex *256) +threadIndex;
  if (index > pixelsAmount -1) {
    return;
  }
  // UINT16 pixelX = index %windowWidth;
  // UINT16 pixelY = index /windowWidth;
  UINT16 pixelY = __umulhi(index ,windowWidthReciprocal); // x *y but returns only high 32 bits, and then get the 16 bits out of the result that i need.
  UINT16 pixelX = index -(pixelY *windowWidth);


  // long pixelIndex = (pixelY *paddedRowBytesAmount) +(pixelX *3);
  // d_bitmap[pixelIndex +0] = rgb.blue; // B
  // d_bitmap[pixelIndex +1] = rgb.green; // G
  // d_bitmap[pixelIndex +2] = rgb.red; // R

  uchar4 color = make_uchar4(rgb.red, rgb.green, rgb.blue, 255);
  surf2Dwrite(color, surface, pixelX *sizeof(uchar4), pixelY);

}