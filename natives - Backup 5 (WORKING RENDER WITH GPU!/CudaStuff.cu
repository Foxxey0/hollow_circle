#include "hip/hip_runtime.h"
#include <stdio.h>
#include <windows.h>

#include "Window.h"
#include "MathHelper.h"

byte * mallocDeviceBitmap(UINT16 windowWidth, UINT16 windowHeight) {
  byte *d_bitmap;
  const UINT32 bytesAmount = rowBytesAmountToPaddedRowBytesAmount(windowWidth *3) *windowHeight;
  hipMalloc(&d_bitmap, bytesAmount);
  return d_bitmap;
}

void copyDeviceBitmapToHostBitmap(byte *bitmap, UINT16 windowWidth, UINT16 windowHeight, byte *d_bitmap) {
  const UINT32 bytesAmount = rowBytesAmountToPaddedRowBytesAmount(windowWidth *3) *windowHeight;
  hipMemcpy(bitmap, d_bitmap, bytesAmount, hipMemcpyDeviceToHost); // to, from
}

void freeDeviceBitmap(byte *d_bitmap) {
  hipFree(d_bitmap);
}



// GPU kernel
__global__ void drawHollowCircleWithBorder_kernel(byte *d_bitmap, UINT16 windowWidth, UINT16 windowHeight, UINT32 pixelsAmount, float circleMiddleX, float circleMiddleY, float radius, float borderWidth, struct colour_RGB rgb) {
  UINT32 threadIndex = (threadIdx.z *blockDim.x *blockDim.y) +(threadIdx.y *blockDim.x) +threadIdx.x;
  UINT32 blockIndex = (blockIdx.z *gridDim.x *gridDim.y) +(blockIdx.y *gridDim.x) +blockIdx.x;
  UINT32 index = blockIndex *blockDim.x *blockDim.y *blockDim.z +threadIndex;
  if (index > pixelsAmount -1) {
    return;
  }
  UINT16 pixelX = index %windowWidth;
  UINT16 pixelY = index /windowWidth;

  float innerRadius = radius -borderWidth;
  if (innerRadius > radius) {
    innerRadius = radius;
  }

  float radiusSquared = radius *radius;
  float innerRadiusSquared = innerRadius *innerRadius;

  float distanceX = pixelX -circleMiddleX;
  float distanceY = pixelY -circleMiddleY;
  float distanceSquared = (distanceX *distanceX) +(distanceY *distanceY);
  if (distanceSquared < radiusSquared && distanceSquared > innerRadiusSquared) {
    // draw white
    long pixelIndex = (pixelY *rowBytesAmountToPaddedRowBytesAmount(windowWidth *3)) +(pixelX *3);
    d_bitmap[pixelIndex +0] = rgb.blue; // B
    d_bitmap[pixelIndex +1] = rgb.green; // G
    d_bitmap[pixelIndex +2] = rgb.red; // R
  } else { // background
    // d_bitmap[(pixelY *rowBytesAmountToPaddedRowBytesAmount(windowWidth *3)) +(pixelX *3 +0)] = 100; // B
    // d_bitmap[(pixelY *rowBytesAmountToPaddedRowBytesAmount(windowWidth *3)) +(pixelX *3 +1)] = 102; // G
    // d_bitmap[(pixelY *rowBytesAmountToPaddedRowBytesAmount(windowWidth *3)) +(pixelX *3 +2)] = 101; // R
  }
}

// extern "C" EXPORT void drawHollowCircleWithBorder(float posX, float posY, float radius, float borderWidth, int width, int height, uint8_t *colours) {
void drawHollowCircleWithBorder(byte *d_bitmap, UINT16 windowWidth, UINT16 windowHeight, float circleMiddleX, float circleMiddleY, float radius, float borderWidth, struct colour_RGB rgb) {
  
  const UINT32 pixelsAmount = windowWidth *windowHeight;

  dim3 gridDim_ = {((pixelsAmount -1) /256) +1, 1, 1};
  dim3 blockDim_ = {32, 4, 2};
  drawHollowCircleWithBorder_kernel<<<gridDim_, blockDim_>>>(d_bitmap, windowWidth, windowHeight, pixelsAmount, circleMiddleX, circleMiddleY, radius, borderWidth, rgb); // 1 thread per pixel

  return;
}



// GPU kernel
__global__ void clearWindow_kernel(byte *d_bitmap, UINT16 windowWidth, UINT16 windowHeight, UINT32 pixelsAmount, struct colour_RGB rgb) {
  UINT32 threadIndex = (threadIdx.z *blockDim.x *blockDim.y) +(threadIdx.y *blockDim.x) +threadIdx.x;
  UINT32 blockIndex = (blockIdx.z *gridDim.x *gridDim.y) +(blockIdx.y *gridDim.x) +blockIdx.x;
  UINT32 index = blockIndex *blockDim.x *blockDim.y *blockDim.z +threadIndex;
  if (index > pixelsAmount -1) {
    return;
  }
  UINT16 pixelX = index %windowWidth;
  UINT16 pixelY = index /windowWidth;


  long pixelIndex = (pixelY *rowBytesAmountToPaddedRowBytesAmount(windowWidth *3)) +(pixelX *3);
  d_bitmap[pixelIndex +0] = rgb.blue; // B
  d_bitmap[pixelIndex +1] = rgb.green; // G
  d_bitmap[pixelIndex +2] = rgb.red; // R

}

void clearWindow(byte *d_bitmap, UINT16 windowWidth, UINT16 windowHeight, struct colour_RGB rgb) {
  
  const UINT32 pixelsAmount = windowWidth *windowHeight;

  dim3 gridDim_ = {((pixelsAmount -1) /256) +1, 1, 1};
  dim3 blockDim_ = {32, 4, 2};
  clearWindow_kernel<<<gridDim_, blockDim_>>>(d_bitmap, windowWidth, windowHeight, pixelsAmount, rgb); // 1 thread per pixel

  return;
}